#include "hip/hip_runtime.h"
/****************************************************************************************
*Reduction kernel that performs add upon first load to shared memory and then unrolls
* the loop when data size is less than 32.
*
*(idea from Nvidia developer zone- CudaGuide- Mark Harris)
*
*
*
*
* ***************************************************************************************/
#define BLOCK_SIZE 1024

//pass as parameters- location in global memory of input data,location where output should be wriiten to
__global__ void reduction_improved(int* g_input,int* g_outputsum){


unsigned int tx= threadIdx.x;
unsigned inti tid=blockIdx.x*(blockDim.x*2)+threadIdx.x; // for block 0 tid will be- 0...1024,for block 1:2048 till 3072. each thread adds two data & loads into shared memory

__shared__ int shared_input[];

//loading data into shared memory

shared_input[tx]=g_input[tid]+g_input[tid+blockDim.x];

__syncthreads();


for(unsigned int stride=blockDim.x/2;strie>32,stride>>=1)
{

if(threadIdx.x<stride)
shared_input[tx]+=shared_input[tx+stride];
__syncthreads();

}

//unrolling last warp here

if(tx<32){
shared_input[tx]+=shared_input[tx+32];
shared_input[tx]+=shared_input[tx+16];
shared_input[tx]+=shared_input[tx+8];
shared_input[tx]+=shared_input[tx+4];
shared_input[tx]+=shared_input[tx+2];
shared_input[tx]+=shared_input[tx+1];


}

//write back result to global memory using thread 0

if(threadIdx.x==0){

g_outputsum[blockIdx.x]=shared_input[0];
}

