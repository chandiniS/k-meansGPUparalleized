#include "hip/hip_runtime.h"
/*****************************************************************************************
*
*				kernelinvoc.cu
*
******************************************************************************************/


#include <stdio.h>
#include <math.h>

#include "kernel.cu"
#include "support.h"

#define BLOCK_SIZE 256
 

void findclusterInvoc(float* xarray, float* yarray, float* cx_h, float* cy_h, float* out_cx, float* out_cy, int numPoints, int numClusters)
{
 float *hx, *hy;
 float *tempx, *tempy;
 float *reducedx, *reducedy;
 int *count_d, *reducCount, *count_h;
 //int num_sec= (numPoints%(BLOCK_SIZE<<1) == 0) ? (numPoints/(BLOCK_SIZE<<1)) : (numPoints/(BLOCK_SIZE<<1) + 1);  
 int num_sec= (numPoints%(BLOCK_SIZE) == 0) ? (numPoints/(BLOCK_SIZE)) : (numPoints/(BLOCK_SIZE) + 1);  
 //int reducedsec= (num_sec%(BLOCK_SIZE<<1) == 0) ? (num_sec/(BLOCK_SIZE<<1)) : (num_sec/(BLOCK_SIZE<<1) + 1);  
 int reducedsec= (num_sec%(BLOCK_SIZE) == 0) ? (num_sec/(BLOCK_SIZE)) : (num_sec/(BLOCK_SIZE) + 1);  
 int clusind = 0;
 float delx, dely;
 bool flag = false;

 dim3 dimGrid, dimGridNext, dimBlock;

  hipError_t cuda_ret;

 /******* Allocating host memory**************************************/

  hx = (float*)malloc(reducedsec*sizeof(float));
  if(hx == NULL)
  {
    printf("Unable to allocate host memory\n");
    exit(0);
  }
  hy = (float*)malloc(reducedsec*sizeof(float));
  if(hy == NULL)
  {
    printf("Unable to allocate host memory\n");
    exit(0);
  }
  count_h = (int*)malloc(reducedsec*sizeof(int));
  if(hy == NULL)
  {
    printf("Unable to allocate host memory\n");
    exit(0);
  }

 /******* Allocating device memory**************************************/

  cuda_ret = hipMemcpyToSymbol(HIP_SYMBOL(cx_cm), cx_h, numClusters*sizeof(float));
  if(cuda_ret != hipSuccess)
  {
    printf("Unable to copy memory to constant memory\n");
    exit(0);
  }
  cuda_ret = hipMemcpyToSymbol(HIP_SYMBOL(cy_cm), cy_h, numClusters*sizeof(float));
  if(cuda_ret != hipSuccess)
  {
    printf("Unable to copy memory to constant memory\n");
    exit(0);
  }

 cuda_ret = hipMalloc((void**)&tempx, num_sec*sizeof(float));
 if(cuda_ret != hipSuccess)
 {
  printf("\n Unable to allocate device memory");
  exit(0);
 }
 cuda_ret = hipMemset(tempx, 0.0, num_sec*sizeof(float));
  if(cuda_ret != hipSuccess)
 {
   printf("\n Unable to allocate device memory");
   exit(0);
 }
 cuda_ret = hipMalloc((void**)&tempy, num_sec*sizeof(float));
  if(cuda_ret != hipSuccess)
 {
   printf("\n Unable to allocate device memory");
   exit(0);
 }
 cuda_ret = hipMemset(tempy, 0.0, num_sec*sizeof(float));
  if(cuda_ret != hipSuccess)
 {
   printf("\n Unable to allocate device memory");
   exit(0);
 }
 cuda_ret = hipMalloc((void**)&count_d, num_sec*sizeof(int));
  if(cuda_ret != hipSuccess)
 {
   printf("\n Unable to allocate device memory");
   exit(0);
 }
 cuda_ret = hipMemset(count_d, 0, num_sec*sizeof(int));
  if(cuda_ret != hipSuccess)
 {
   printf("\n Unable to allocate device memory");
   exit(0);
 }


 cuda_ret = hipMalloc((void**)&reducedx, reducedsec*sizeof(float));
 if(cuda_ret != hipSuccess)
 {
  printf("\n Unable to allocate device memory");
  exit(0);
 }
 cuda_ret = hipMemset(reducedx, 0.0, reducedsec*sizeof(float));
  if(cuda_ret != hipSuccess)
 {
   printf("\n Unable to allocate device memory");
   exit(0);
 }
 cuda_ret = hipMalloc((void**)&reducedy, reducedsec*sizeof(float));
 if(cuda_ret != hipSuccess)
 {
  printf("\n Unable to allocate device memory");
  exit(0);
 }
 cuda_ret = hipMemset(reducedy, 0.0, reducedsec*sizeof(float));
  if(cuda_ret != hipSuccess)
 {
   printf("\n Unable to allocate device memory");
   exit(0);
 }
 cuda_ret = hipMalloc((void**)&reducCount, reducedsec*sizeof(int));
 if(cuda_ret != hipSuccess)
 {
  printf("\n Unable to allocate device memory");
  exit(0);
 }
 cuda_ret = hipMemset(reducCount, 0, reducedsec*sizeof(int));
  if(cuda_ret != hipSuccess)
 {
   printf("\n Unable to allocate device memory");
   exit(0);
 }

 hipDeviceSynchronize();

 /***************Invoking kernel*****************************************/

  dimGrid = num_sec;
  dimGridNext = reducedsec;
  dimBlock = BLOCK_SIZE;

  int loop = 0;
 
   do
   { 
    flag = false;
    ++loop;
    clusind = 0;

    while(clusind < numClusters)
   {
     findCluster<<<dimGrid, dimBlock>>>(xarray, yarray, tempx, tempy, count_d, numPoints, numClusters, clusind);

     reduceResult<<<dimGridNext, dimBlock>>>(tempx, tempy, count_d, reducedx, reducedy, reducCount, num_sec);
     
     cuda_ret = hipMemcpy(hx, reducedx, reducedsec*sizeof(float), hipMemcpyDeviceToHost);
     if(cuda_ret != hipSuccess)
     {
       printf("Unable to copy device to host");
       exit(0);
     }
     cuda_ret = hipMemcpy(hy, reducedy, reducedsec*sizeof(float), hipMemcpyDeviceToHost);
     if(cuda_ret != hipSuccess)
     {
       printf("Unable to copy device to host");
       exit(0);
     }
     cuda_ret = hipMemcpy(count_h, reducCount, reducedsec*sizeof(int), hipMemcpyDeviceToHost);
     if(cuda_ret != hipSuccess)
     {
       printf("Unable to copy device to host");
       exit(0);
     }
     
     hipDeviceSynchronize();
     
    // printf("loop = %d, clusind = %d, sumx = %f, sumy = %f, num = %d\n",loop, clusind, hx[0], hy[0], count_h[0]);
     hx[0] /= count_h[0];
     hy[0] /= count_h[0];
     out_cx[clusind] = hx[0];
     out_cy[clusind] = hy[0];
     clusind++;
   }

    for(int i = 0; i < numClusters; ++i)
    {
     delx = (cx_h[i] - out_cx[i])<0 ? (out_cx[i]-cx_h[i]) : (cx_h[i] - out_cx[i]);
     dely = (cy_h[i] - out_cy[i])<0 ? (out_cy[i]-cy_h[i]) : (cy_h[i] - out_cy[i]);
     if(delx > pow(10, -2) || dely > pow(10, -2))
     {
       flag = true;
     }
     //printf("delx = %f, dely = %f\n", delx, dely);
    }

    // Checking cluster centers values --------------------------------------

   //printf("Initial and current values\n");

   for(int j = 0; j < numClusters; ++j)
   {
     //printf("For cluster %d :-\n",j+1);
     //printf("x-> ini = %f, cur = %f\n", cx_h[j], out_cx[j]);
     //printf("y-> ini = %f, cur = %f\n", cy_h[j], out_cy[j]);
   }

    
    if(flag == true)
    {
      memcpy(cx_h, out_cx, numClusters*sizeof(float));
      memcpy(cy_h, out_cy, numClusters*sizeof(float));

     cuda_ret = hipMemcpyToSymbol(HIP_SYMBOL(cx_cm), out_cx, numClusters*sizeof(float));
     if(cuda_ret != hipSuccess)
     {
      printf("\n Unable to copy to constant memory");
      exit(0);
     }
     cuda_ret = hipMemcpyToSymbol(HIP_SYMBOL(cy_cm), out_cy, numClusters*sizeof(float));
     if(cuda_ret != hipSuccess)
     {
      printf("\n Unable to copy to constant memory");
      exit(0);
     }

     cuda_ret = hipMemset(tempx, 0.0, num_sec*sizeof(float));
     if(cuda_ret != hipSuccess)
     {
       printf("\n Unable to allocate device memory");
       exit(0);
     }
     cuda_ret = hipMemset(tempy, 0.0, num_sec*sizeof(float));
     if(cuda_ret != hipSuccess)
     {
       printf("\n Unable to allocate device memory");
       exit(0);
     }
     cuda_ret = hipMemset(count_d, 0, num_sec*sizeof(int));
     if(cuda_ret != hipSuccess)
     {
       printf("\n Unable to allocate device memory");
       exit(0);
     }
     cuda_ret = hipMemset(reducedx, 0.0, reducedsec*sizeof(float));
     if(cuda_ret != hipSuccess)
     {
       printf("\n Unable to allocate device memory");
       exit(0);
     }
     cuda_ret = hipMemset(reducedy, 0.0, reducedsec*sizeof(float));
     if(cuda_ret != hipSuccess)
     {
       printf("\n Unable to allocate device memory");
       exit(0);
     }
     cuda_ret = hipMemset(reducCount, 0, reducedsec*sizeof(int));
     if(cuda_ret != hipSuccess)
     {
       printf("\n Unable to allocate device memory");
       exit(0);
     }


     hipDeviceSynchronize();
    }

    if(loop == 2)
    {
    // break;
    }
    
  }while(flag == true);
 
   // Free Memory -------------------------------------------------------------

   hipFree(tempx);
   hipFree(tempy);
   hipFree(count_d);
   hipFree(reducedx);
   hipFree(reducedy);
   hipFree(reducCount);
   free(hx);
   free(hy);
   free(count_h);
 
 return;
}
