/*****************************************************************************************************
*
*					main.cu
*
*****************************************************************************************************/

#include <stdio.h>

#include "support.h"

int main(int argc, char* argv[])
{
 Timer timer;

 //	Initialize variables -----------------------------------------------------

 float *xarr_h, *yarr_h;
 float *xarr_d, *yarr_d;
 float *cx_h, *cy_h;
 float *out_cx, *out_cy;
 int numPoints, numClusters;
 char *datafile;
 hipError_t cuda_ret;

 //  Allocate values to variables through arguments
 
 printf("\n Setting up the problem ........");
 fflush(stdout);
 startTime(&timer);

 if(argc == 1)
 {
   datafile = "color_histogram68040.txt";
   numClusters = 11;
 }
 else if(argc == 2)
 {
  /*if(sscanf(argv[1], "%s", &datafile) != 1)
  { 
    printf("\nArgument is not an integer");
    exit(0);
  }*/
  datafile = argv[1];
   numClusters = 11;
 }
 else if(argc == 3)
 {
  /*if(sscanf(argv[1], "%s", &datafile) != 1)
  {
   printf("\n Argument for file of points is not a string");
   exit(0);
  }*/
  datafile = argv[1];
  if(sscanf(argv[2], "%d", &numClusters) != 1)
  {
   printf("\n Argument for number of clusters is not an integer");
   exit(0);
  }
 }
 else 
 {
   printf("\n Invalid input parameters");
   exit(0);
 }
 stopTime(&timer);
 printf("%f s\n", elapsedTime(timer));
  

 // Number of points from the input file  ---------------------------------------

 numPoints = countPoints(datafile); 
 printf("\nInput size for points = %d, Size for clusters = %d\n", numPoints, numClusters);

 // Allocating and initializing host variables -------------------------------

  printf("Allocating host variables ................");
  fflush(stdout);
  startTime(&timer);  

  xarr_h = (float*)malloc(numPoints * sizeof(float));
  if(xarr_h == NULL)
  {
   printf("\n Unable to allcate host variable");
   exit(0);
  } 
  yarr_h = (float*)malloc(numPoints * sizeof(float));
  if(yarr_h == NULL)
  {
    printf("\n Unable to allocate host variable");
    exit(0);
  }
  cx_h = (float*)malloc(numClusters * sizeof(float));
  if(cx_h == NULL)
  {
    printf("\n Unable to allocate host variables");
    exit(0);
  }
  cy_h = (float*)malloc(numClusters * sizeof(float));
  if(cy_h == NULL)
  {
    printf("\n Unable to allocate host variables");
    exit(0);
  }
  out_cx = (float*)malloc(numClusters * sizeof(float));
  if(out_cx == NULL)
  {
    printf("\n Unable to allocate host variables");
    exit(0);
  }
  out_cy = (float*)malloc(numClusters * sizeof(float));
  if(out_cy == NULL)
  {
    printf("\n Unable to allocate host variables");
    exit(0);
  }

  stopTime(&timer);
  printf("%f s\n", elapsedTime(timer));

 // Allocating device variables ----------------------------------------------

 printf("\n Allocating device variables........");
 fflush(stdout);
 startTime(&timer);

 cuda_ret = hipMalloc((void**)&xarr_d, numPoints*sizeof(float));
 if(cuda_ret != hipSuccess)
 {
   printf("\n Unable to allocate device memory");
   exit(0);
 }
 cuda_ret = hipMalloc((void**)&yarr_d, numPoints*sizeof(float));
 if(cuda_ret != hipSuccess)
 {
   printf("\n Unable to allocate device memory");
   exit(0);
 }
 
 hipDeviceSynchronize();
 stopTime(&timer);
 printf("%f s\n", elapsedTime(timer));
 
 
 // Random generation of points in 2-D plane  ---------------------------------------
 
  //observationGen(numPoints);

 // Store Points in host variables   -----------------------------------------------

  storePoints(xarr_h, yarr_h, datafile, numPoints);

 // Randomly select distinct numClusters points from availabe points -----------------

  clusterPoints(cx_h, cy_h, xarr_h, yarr_h, numClusters, numPoints);
   // Initial cluster centers values --------------------------------------

  /* printf("Initial values for cluster centres\n");

   for(int j = 0; j < numClusters; ++j)
   {
     printf("For cluster %d :-\n",j+1);
     printf("x-> ini = %f\n", cx_h[j]);
     printf("y-> ini = %f\n", cy_h[j]);
   }
*/
  // Copy host variables to device  memory ---------------------------------------------------

  printf("\nCopying data from host to device\n");
  fflush(stdout);
  startTime(&timer);

  cuda_ret = hipMemcpy(xarr_d, xarr_h, numPoints*sizeof(float), hipMemcpyHostToDevice);
  if(cuda_ret != hipSuccess)
  {
    printf("Unable to copy memory to device\n");
    exit(0);
  }
  cuda_ret = hipMemcpy(yarr_d, yarr_h, numPoints*sizeof(float), hipMemcpyHostToDevice);
  if(cuda_ret != hipSuccess)
  {
    printf("Unable to copy memory to device\n");
    exit(0);
  }
  
  hipDeviceSynchronize();
  stopTime(&timer);
  printf("%f s\n", elapsedTime(timer)); 

  // Kernel invocation

   /*printf("Launching kernel ...........\n");
   fflush(stdout);
   startTime(&timer);*/

   /* Kernel will get setup and invoked inside findclusterInvok function in kernelinvoc.cu file*/

   findclusterInvoc(xarr_d, yarr_d, cx_h, cy_h, out_cx, out_cy, numPoints, numClusters);

   cuda_ret = hipDeviceSynchronize();
   if(cuda_ret != hipSuccess)
   {
     printf("Unable to launch/execute kernel\n");
     exit(0);
   }

   // Checking cluster centers values --------------------------------------

/*   printf("Initial and final values\n");

   for(int j = 0; j < numClusters; ++j)
   {
     printf("For cluster %d :-\n",j+1);
     printf("x-> ini = %f, fin = %f\n", cx_h[j], out_cx[j]);
     printf("y-> ini = %f, fin = %f\n", cy_h[j], out_cy[j]);
   }
*/
//   stopTime(&timer);
  // printf("Elapsed time for kernel execution = %f s\n", elapsedTime(timer));

  //Getting cluster centers in file outCenter.txt

   clusterCenters(out_cx, out_cy, numClusters);


  // Free Memory ----------------------------------------------------

  hipFree(xarr_d);
  hipFree(yarr_d);
  hipFree(out_cx);
  hipFree(out_cy);
  free(xarr_h);
  free(yarr_h);
  free(cx_h);
  free(cy_h);
 
 return 0;
}
