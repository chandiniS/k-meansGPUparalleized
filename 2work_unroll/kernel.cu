#include "hip/hip_runtime.h"
/*******************************************************************************************
*
*      				          kernel.cu
*
*******************************************************************************************/

#include <math.h>

#include "support.h"

#define BLOCK_SIZE 256

__global__ void findCluster(float* xarray, float* yarray, float* tempx, float* tempy, int* countP, int numPoints, int numClusters, int clusid)
{
 
 __shared__ float sx[2*BLOCK_SIZE];
 __shared__ float sy[2*BLOCK_SIZE];
 __shared__ int cp[2*BLOCK_SIZE];
 
 //int i = blockIdx.x * blockDim.x + threadIdx.x;
 int startSection = 2*blockIdx.x*blockDim.x;
 int index1 = startSection + threadIdx.x;
 int index2 = startSection + blockDim.x + threadIdx.x;

 sx[threadIdx.x] = (index1 < numPoints) ? xarray[index1] : 0.0;
 sx[blockDim.x + threadIdx.x] = (index2 < numPoints) ? xarray[index2] : 0.0;
 sy[threadIdx.x] = (index1 < numPoints) ? yarray[index1] : 0.0;
 sy[blockDim.x + threadIdx.x] = (index2 < numPoints) ? yarray[index2] : 0.0;
 cp[threadIdx.x] = (index1 < numPoints) ? 1 : 0;
 cp[blockDim.x + threadIdx.x] = (index2 < numPoints) ? 1 : 0;

 __syncthreads();
  
  // Finding points that belongs to this cluster


 int id = 0;
 float distMin = (sx[threadIdx.x] - cx_cm[id])*(sx[threadIdx.x] - cx_cm[id]) + (sy[threadIdx.x] - cy_cm[id])*(sy[threadIdx.x] - cy_cm[id]);

 float distTemp;
 for(int k = 1; k < numClusters; ++k) 
 {
   distTemp = (sx[threadIdx.x] - cx_cm[k])*(sx[threadIdx.x] - cx_cm[k]) + (sy[threadIdx.x] - cy_cm[k])*(sy[threadIdx.x] - cy_cm[k]);
   if(distTemp < distMin)
   {
     distMin = distTemp;
     id = k;
   }
 }

 if (id != clusid)
 {
   sx[threadIdx.x] = 0.0;
   sy[threadIdx.x] = 0.0;
   cp[threadIdx.x] = 0;
 }

 id = 0;
 distMin = (sx[blockDim.x + threadIdx.x] - cx_cm[id])*(sx[blockDim.x + threadIdx.x] - cx_cm[id]) + (sy[blockDim.x + threadIdx.x] - cy_cm[id])*(sy[blockDim.x + threadIdx.x] - cy_cm[id]);
  
 for(int k = 1; k < numClusters; ++k)
 {
  distTemp = (sx[blockDim.x + threadIdx.x] - cx_cm[k])*(sx[blockDim.x + threadIdx.x] - cx_cm[k]) + (sy[blockDim.x + threadIdx.x] - cy_cm[k])*(sy[blockDim.x + threadIdx.x] - cy_cm[k]);
   if(distTemp < distMin)
   {
     distMin = distTemp;
     id = k;
   }
 }
 
 if (id != clusid)
 {
   sx[blockDim.x + threadIdx.x] = 0.0;
   sy[blockDim.x + threadIdx.x] = 0.0;
   cp[blockDim.x + threadIdx.x] = 0;
 }
 //    printf("cp[%d] = %d\n", threadIdx.x, cp[threadIdx.x]);
 // Summing the value of points and number of points with the help of reduction
 
 for(int stride = blockDim.x; stride > 32; stride >>= 1) 
 {
   __syncthreads();
    
   if(threadIdx.x < stride)
   {
     sx[threadIdx.x] += sx[stride + threadIdx.x];
     sy[threadIdx.x] += sy[stride + threadIdx.x];
     cp[threadIdx.x] += cp[stride + threadIdx.x];
  //   printf("cp[%d] = %d\n", threadIdx.x, cp[threadIdx.x]);
   }

 }

//unrolling last warp here

   if(threadIdx.x < 32)
   {
    __syncthreads();
     
     sx[threadIdx.x] += sx[32 + threadIdx.x];
     sy[threadIdx.x] += sy[32 + threadIdx.x];
     cp[threadIdx.x] += cp[32 + threadIdx.x];
    __syncthreads();
     sx[threadIdx.x] += sx[16 + threadIdx.x];
     sy[threadIdx.x] += sy[16 + threadIdx.x];
     cp[threadIdx.x] += cp[16 + threadIdx.x];
    __syncthreads();
     sx[threadIdx.x] += sx[8 + threadIdx.x];
     sy[threadIdx.x] += sy[8 + threadIdx.x];
     cp[threadIdx.x] += cp[8 + threadIdx.x];
    __syncthreads();
     sx[threadIdx.x] += sx[4 + threadIdx.x];
     sy[threadIdx.x] += sy[4 + threadIdx.x];
     cp[threadIdx.x] += cp[4 + threadIdx.x];
    __syncthreads();
     sx[threadIdx.x] += sx[2 + threadIdx.x];
     sy[threadIdx.x] += sy[2 + threadIdx.x];
     cp[threadIdx.x] += cp[2 + threadIdx.x];
    __syncthreads();
     sx[threadIdx.x] += sx[1 + threadIdx.x];
     sy[threadIdx.x] += sy[1 + threadIdx.x];
     cp[threadIdx.x] += cp[1 + threadIdx.x];
   }

   __syncthreads();
 tempx[blockIdx.x] = sx[0]; 
 tempy[blockIdx.x] = sy[0]; 
 countP[blockIdx.x] = cp[0];
}


__global__ void reduceResult(float* tempx, float* tempy, int* count_d, float* reducedx, float* reducedy, int* reducCount, int num_sec)
{
 __shared__ float sumx[2*BLOCK_SIZE];
 __shared__ float sumy[2*BLOCK_SIZE];
 __shared__ int sumcount[2*BLOCK_SIZE];

    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;

    unsigned int indexin = start + t;
    unsigned int indexpast = start + blockDim.x + t;
    sumx[t] = (indexin < num_sec) ? tempx[indexin] : 0.0;
    sumx[blockDim.x + t] = (indexpast < num_sec) ? tempx[indexpast] : 0.0;
    sumy[t] = (indexin < num_sec) ? tempy[indexin] : 0.0;
    sumy[blockDim.x + t] = (indexpast < num_sec) ? tempy[indexpast] : 0.0;
    sumcount[t] = (indexin < num_sec) ? count_d[indexin] : 0;
    sumcount[blockDim.x + t] = (indexpast < num_sec) ? count_d[indexpast] : 0;
    
    for (unsigned int stride = blockDim.x; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (t < stride)
        {
          sumx[t] += sumx[t + stride];
          sumy[t] += sumy[t + stride];
          sumcount[t] += sumcount[t + stride];
        }
    }
    reducedx[blockIdx.x] = sumx[0];
    reducedy[blockIdx.x] = sumy[0];
    reducCount[blockIdx.x] = sumcount[0];
}
